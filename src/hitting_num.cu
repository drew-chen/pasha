#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "hitting_num.cuh"

using namespace std;
using unsigned_int = uint64_t;
using byte = uint8_t;

#define NUM_THREADS 256


__constant__ unsigned_int d_vertexExp;
unsigned_int vertexExp;
unsigned_int L;
unsigned_int dSize;
// total number of edges before any removal (does not change)
unsigned_int numEdges;

// L-k+1 rows, vertexExp columns
byte* edgeArray_gpu;
float* Fprev_gpu;
float* Fcurr_gpu;
// a host pointer pointing to the copy of D on the gpu
float* D_gpu;


// assumes already inited
__device__ float D_get(float* D, int row, int col) {
    return D[row*d_vertexExp + col];
}
__device__ void D_set(float* D, int row, int col, float val) {
    D[row*d_vertexExp + col] = val;
}

void initHittingNum(unsigned_int LParam, unsigned_int vertexExpParam, unsigned_int dSizeParam, unsigned_int numEdgesParam, byte* edgeArray) {
    L = LParam;
    vertexExp = vertexExpParam;
    dSize = dSizeParam;
    numEdges = numEdgesParam;

    hipMalloc((void**)&edgeArray_gpu, numEdges*sizeof(byte)); 
    hipMalloc((void**)&D_gpu, dSize*sizeof(float));
    hipMalloc((void**)&Fprev_gpu, vertexExp*sizeof(float));
    hipMalloc((void**)&Fcurr_gpu, vertexExp*sizeof(float));

    // MemcpyToSymbol is for consts
    hipMemcpyToSymbol(HIP_SYMBOL(d_vertexExp), &vertexExpParam, sizeof(unsigned_int));
}

void finalizeHittingNum() {
    hipFree(edgeArray_gpu);
    hipFree(D_gpu);
    hipFree(Fprev_gpu);
    hipFree(Fcurr_gpu);
}

__global__ void setInitialDFprev_gpu(float* D, float* Fprev) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= d_vertexExp) return;
    D_set(D, 0, tid, 1.4e-45);
    Fprev[tid] = 1.4e-45;
}

__global__ void calcNumStartingPathsOneIter_gpu(float* D, byte* edgeArray, int j) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= d_vertexExp) return;
    unsigned_int vertexExp2 = d_vertexExp * 2;
    unsigned_int vertexExp3 = d_vertexExp * 3;
    
    D_set(D, j, i, 
        edgeArray[i]*D_get(D, j-1, (i >> 2))
            + edgeArray[i + d_vertexExp]*D_get(D, j-1,((i + d_vertexExp) >> 2))
            + edgeArray[i + vertexExp2]*D_get(D, j-1,((i + vertexExp2) >> 2))
            + edgeArray[i + vertexExp3]*D_get(D, j-1,((i + vertexExp3) >> 2))
    );
}

void calcNumStartingPaths(byte* edgeArray, float* D, float* Fprev) {
    /**
    * This function generates D. D(v,i): # of i long paths starting from v after decycling
    */
    // want tid range [0, vertexExp)
    // edgeArray changes outside of this func so must cpy
    hipMemcpy(edgeArray_gpu, edgeArray, numEdges*sizeof(byte), hipMemcpyHostToDevice);

    int grid_size = 1 + ((vertexExp - 1) / NUM_THREADS);
    setInitialDFprev_gpu<<<grid_size, NUM_THREADS>>>(D_gpu, Fprev_gpu); 

    // THE ISSUE IS HERE
    // // TODO: replace loop with this https://towardsdatascience.com/gpu-optimized-dynamic-programming-8d5ba3d7064f
    for (unsigned_int j = 1; j <= L; j++) {
        calcNumStartingPathsOneIter_gpu<<<grid_size, NUM_THREADS>>>(D_gpu, edgeArray_gpu, j); 
    }



    hipMemcpy(D, D_gpu, dSize*sizeof(float),  hipMemcpyDeviceToHost);
    hipMemcpy(Fprev, Fprev_gpu, vertexExp*sizeof(float),  hipMemcpyDeviceToHost);
    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) 
    //     printf("Error: %s\n", hipGetErrorString(err));
    // printf("---END---\n");
}
